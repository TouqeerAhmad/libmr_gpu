#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


__global__
void normalization_and_sum(int size, double maxx, double range, double *inputArr, double *x0, double *mean)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size)
  {
    x0[i] = (inputArr[i] - maxx) / range;
    atomicAdd(mean,x0[i]);
  }
}


__global__
void compute_std(int size, double *x0, double *mean, double *myStd)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  double tempVal, tempValSq;
  if (i < size)
  {
    tempVal = x0[i] - *(mean);
    tempValSq = tempVal * tempVal;
    atomicAdd(myStd,tempValSq);
  }
}


__global__
void scale_likelihood(int size, double sigma, double *sumxw, double *sumw, double *x, double *w)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  double wLocal;
  if (i < size)
  {
    wLocal = w[i] * exp(x[i]/sigma);
    atomicAdd(sumw,wLocal);
    atomicAdd(sumxw,wLocal*x[i]);
  }
}




#ifdef __cplusplus
extern "C" {
#endif

void runKernels()
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  
  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}



double runKernels_ScaleLikelihood(double sigma, double *x, double *w, double xbar, int size)
{
  
  double sumxw = 0.0;
  double sumw = 0.0; 
  
  double *device_x, *device_w, *device_sumxw, *device_sumw;
  
  hipMalloc(&device_x, size*sizeof(double)); 
  hipMalloc(&device_w, size*sizeof(double));
  hipMalloc(&device_sumxw, sizeof(double));
  hipMalloc(&device_sumw, sizeof(double));
  
  hipMemcpy(device_x, x, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_w, w, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_sumxw, &sumxw, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_sumw, &sumw, sizeof(double), hipMemcpyHostToDevice);
  
  scale_likelihood<<<(size+255)/256, 256>>>(size, sigma, device_sumxw, device_sumw, device_x, device_w);
  
  // copy the sums to host
  hipMemcpy(&sumxw, device_sumxw, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&sumw, device_sumw, sizeof(double), hipMemcpyDeviceToHost);
  
  
  double v;
  v = (sigma + xbar - sumxw / sumw);
  
  hipFree(device_sumw);
  hipFree(device_sumxw);
  hipFree(device_w);
  hipFree(device_x);
  
  return v;
}


void runKernels_ComputeMeanAndStd(double *inputData, double *x0, double *mean, double *myStd, double maxx, double range, int size)
{
  
  double *device_inputData, *device_x0, *device_mean, *device_myStd;
  
  hipMalloc(&device_inputData, size*sizeof(double)); 
  hipMalloc(&device_x0, size*sizeof(double));
  hipMalloc(&device_mean, sizeof(double));
  hipMalloc(&device_myStd, sizeof(double));
  
  
  hipMemcpy(device_inputData, inputData, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_x0, x0, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_myStd, myStd, sizeof(double), hipMemcpyHostToDevice);
  
  
  normalization_and_sum<<<(size+255)/256, 256>>>(size, maxx, range, device_inputData, device_x0, device_mean);
  
  // copy the sum to host
  hipMemcpy(mean, device_mean, sizeof(double), hipMemcpyDeviceToHost);
  
  // compute the mean 
  *(mean) = *(mean) / size; 
  
  // copy mean to the device
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  
  
  compute_std<<<(size+255)/256, 256>>>(size, device_x0, device_mean, device_myStd);
  
  // copy the sum to host
  hipMemcpy(myStd, device_myStd, sizeof(double), hipMemcpyDeviceToHost);
  
  *(myStd) = *(myStd) / (size-1);
  *(myStd) = sqrt(*(myStd));
      
  // copy x0 to host
  hipMemcpy(x0, device_x0, size*sizeof(double), hipMemcpyDeviceToHost);
  
  
  hipFree(device_myStd);
  hipFree(device_mean);
  hipFree(device_x0);
  hipFree(device_inputData);
  
}


#ifdef __cplusplus
}
#endif

