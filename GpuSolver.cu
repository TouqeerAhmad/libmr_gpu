#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


__global__
void normalization_and_sum(int size, double maxx, double range, double *inputArr, double *x0, double *mean)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size)
  {
    x0[i] = (inputArr[i] - maxx) / range;
    atomicAdd(mean,x0[i]);
  }
}


__global__
void compute_std(int size, double *x0, double *mean, double *myStd)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  double tempVal, tempValSq;
  if (i < size)
  {
    tempVal = x0[i] - *(mean);
    tempValSq = tempVal * tempVal;
    atomicAdd(myStd,tempValSq);
  }
}



#ifdef __cplusplus
extern "C" {
#endif

void runKernels()
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  
  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}


void runKernels_ComputeMeanAndStd(double * inputData, double * x0, double *mean, double *myStd, double maxx, double range, int size)
{
  
  double *device_inputData, *device_x0, *device_mean, *device_myStd;
  
  hipMalloc(&device_inputData, size*sizeof(double)); 
  hipMalloc(&device_x0, size*sizeof(double));
  hipMalloc(&device_mean, sizeof(double));
  hipMalloc(&device_myStd, sizeof(double));
  
  
  hipMemcpy(device_inputData, inputData, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_x0, x0, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_myStd, myStd, sizeof(double), hipMemcpyHostToDevice);
  
  
  normalization_and_sum<<<(size+255)/256, 256>>>(size, maxx, range, device_inputData, device_x0, device_mean);
  
  // copy the sum to host
  hipMemcpy(mean, device_mean, sizeof(double), hipMemcpyDeviceToHost);
  
  // compute the mean 
  *(mean) = *(mean) / size; 
  
  // copy mean to the device
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  
  
  compute_std<<<(size+255)/256, 256>>>(size, device_x0, device_mean, device_myStd);
  
  // copy the sum to host
  hipMemcpy(myStd, device_myStd, sizeof(double), hipMemcpyDeviceToHost);
  
  *(myStd) = *(myStd) / (size-1);
  *(myStd) = sqrt(*(myStd));
      
  // copy x0 to host
  hipMemcpy(x0, device_x0, size*sizeof(double), hipMemcpyDeviceToHost);
  
  
  hipFree(device_myStd);
  hipFree(device_mean);
  hipFree(device_x0);
  hipFree(device_inputData);
  
}


#ifdef __cplusplus
}
#endif

