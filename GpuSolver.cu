#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


__global__
void normalization_and_sum(int size, double maxx, double range, double *inputArr, double *x0, double *mean)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size)
  {
    x0[i] = (inputArr[i] - maxx) / range;
    atomicAdd(mean,x0[i]);
  }
}


__global__
void compute_std(int size, double *x0, double *mean, double *myStd)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  double tempVal, tempValSq;
  if (i < size)
  {
    tempVal = x0[i] - *(mean);
    tempValSq = tempVal * tempVal;
    atomicAdd(myStd,tempValSq);
  }
}


__global__
void scale_likelihood(int size, double sigma, double *sumxw, double *sumw, double *x, double *w)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  double wLocal;
  if (i < size)
  {
    wLocal = w[i] * exp(x[i]/sigma);
    atomicAdd(sumw,wLocal);
    atomicAdd(sumxw,wLocal*x[i]);
  }
}


__global__
void neg_log_likelihood(double size, double mu, double sigma, double logSigma, double *data, double *censoring, double *frequency, double *nH11, double *nH12, double *nH22, double *nlogL)
{
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < size)
  {
    double z, expz, L, unc;
    
    z = (data[i]-mu)/sigma;
    expz = exp(z);
    L = (z-logSigma)*(1-censoring[i]-expz);
    unc = (1-censoring[i]);
    
    atomicAdd(nlogL,frequency[i]*L);
    atomicAdd(nH11,frequency[i]*expz);
    atomicAdd(nH12, frequency[i] * ((z + 1) * expz - unc));
    atomicAdd(nH22, frequency[i] * (z *(z + 2) * expz - ((2 * z + 1) *unc)));
  }
}


#ifdef __cplusplus
extern "C" {
#endif

void runKernels()
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));
  
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  
  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}


void runKernels_NegLogLikelihood(double* nlogL, double* acov, double* weibulparms, double* data, double* censoring, double* frequency, int size)
{

  double mu = weibulparms[0]; // scale
  double sigma = weibulparms[1]; // shape
  double logSigma;
  
  
  double nH11 = 0.0;
  double nH12 = 0.0;
  double nH22 = 0.0;
  
  logSigma = log(sigma);
  
  double *dev_data, *dev_censoring, *dev_frequency, *dev_nH11, *dev_nH12, *dev_nH22, *dev_nlogL;
  
  hipMalloc(&dev_data, size*sizeof(double));
  hipMalloc(&dev_censoring, size*sizeof(double));
  hipMalloc(&dev_frequency, size*sizeof(double));
  hipMalloc(&dev_nH11, sizeof(double));
  hipMalloc(&dev_nH12, sizeof(double));
  hipMalloc(&dev_nH22, sizeof(double));
  hipMalloc(&dev_nlogL, sizeof(double));
  
  hipMemcpy(dev_data, data, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_censoring, censoring, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_frequency, frequency, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_nH11, &nH11, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_nH12, &nH12, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_nH22, &nH22, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_nlogL, nlogL, sizeof(double), hipMemcpyHostToDevice);
  
  neg_log_likelihood<<<(size+255)/256, 256>>>(size, mu, sigma, logSigma, dev_data, dev_censoring, dev_frequency, dev_nH11, dev_nH12, dev_nH22, dev_nlogL);
  
  // copy to host
  hipMemcpy(nlogL, dev_nlogL, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&nH11, dev_nH11, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&nH12, dev_nH12, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&nH22, dev_nH22, sizeof(double), hipMemcpyDeviceToHost);
  
  *nlogL = *nlogL * -1;

  double sigmaSq = sigma * sigma;
  double avarDenom = (nH11*nH22 - nH12*nH12);
      
  printf("avarDenom gpu %f\n", avarDenom);
    
  acov[0]=sigmaSq*(nH22/avarDenom);
  acov[1]=sigmaSq*((-1*nH12)/avarDenom);
  acov[2]=sigmaSq*((-1*nH12)/avarDenom);
  acov[3]=sigmaSq*(nH11/avarDenom);

  hipFree(dev_nlogL);
  hipFree(dev_nH22);
  hipFree(dev_nH12);
  hipFree(dev_nH11);
  hipFree(dev_frequency);
  hipFree(dev_censoring);
  hipFree(dev_data);
  
}

double runKernels_ScaleLikelihood(double sigma, double *x, double *w, double xbar, int size)
{
  
  double sumxw = 0.0;
  double sumw = 0.0; 
  
  double *device_x, *device_w, *device_sumxw, *device_sumw;
  
  hipMalloc(&device_x, size*sizeof(double)); 
  hipMalloc(&device_w, size*sizeof(double));
  hipMalloc(&device_sumxw, sizeof(double));
  hipMalloc(&device_sumw, sizeof(double));
  
  hipMemcpy(device_x, x, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_w, w, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_sumxw, &sumxw, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_sumw, &sumw, sizeof(double), hipMemcpyHostToDevice);
  
  scale_likelihood<<<(size+255)/256, 256>>>(size, sigma, device_sumxw, device_sumw, device_x, device_w);
  
  // copy the sums to host
  hipMemcpy(&sumxw, device_sumxw, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&sumw, device_sumw, sizeof(double), hipMemcpyDeviceToHost);
  
  
  double v;
  v = (sigma + xbar - sumxw / sumw);
  
  hipFree(device_sumw);
  hipFree(device_sumxw);
  hipFree(device_w);
  hipFree(device_x);
  
  return v;
}


void runKernels_ComputeMeanAndStd(double *inputData, double *x0, double *mean, double *myStd, double maxx, double range, int size)
{
  
  double *device_inputData, *device_x0, *device_mean, *device_myStd;
  
  hipMalloc(&device_inputData, size*sizeof(double)); 
  hipMalloc(&device_x0, size*sizeof(double));
  hipMalloc(&device_mean, sizeof(double));
  hipMalloc(&device_myStd, sizeof(double));
  
  
  hipMemcpy(device_inputData, inputData, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_x0, x0, size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_myStd, myStd, sizeof(double), hipMemcpyHostToDevice);
  
  
  normalization_and_sum<<<(size+255)/256, 256>>>(size, maxx, range, device_inputData, device_x0, device_mean);
  
  // copy the sum to host
  hipMemcpy(mean, device_mean, sizeof(double), hipMemcpyDeviceToHost);
  
  // compute the mean 
  *(mean) = *(mean) / size; 
  
  // copy mean to the device
  hipMemcpy(device_mean, mean, sizeof(double), hipMemcpyHostToDevice);
  
  
  compute_std<<<(size+255)/256, 256>>>(size, device_x0, device_mean, device_myStd);
  
  // copy the sum to host
  hipMemcpy(myStd, device_myStd, sizeof(double), hipMemcpyDeviceToHost);
  
  *(myStd) = *(myStd) / (size-1);
  *(myStd) = sqrt(*(myStd));
      
  // copy x0 to host
  hipMemcpy(x0, device_x0, size*sizeof(double), hipMemcpyDeviceToHost);
  
  
  hipFree(device_myStd);
  hipFree(device_mean);
  hipFree(device_x0);
  hipFree(device_inputData);
  
}


#ifdef __cplusplus
}
#endif

